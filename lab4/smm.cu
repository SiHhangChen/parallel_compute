#include <iostream>
#include <random>
#include <string>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void smm(int* denseMatrix, int* sparseMatrixIdx, int* sparseMatrixVal, int* result, int M, int N, int P, int K) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int sum;
    if (idx > K) return;
    else {
        int row = sparseMatrixIdx[idx * 2];
        int col = sparseMatrixIdx[idx * 2 + 1];
        int value = sparseMatrixVal[idx];
        for(int i = 0; i < M; i++) {
            // result[i * P + col] += denseMatrix[i * N + row] * value;
            sum += denseMatrix[i * N + row] * value;
            // atomicAdd(&result[i * P + col], denseMatrix[i * N + row] * value);
        }
        result[row * P + col] = sum;
    }
    // __syncthreads();
}

int main() {
    srand(time(NULL));
    // 这里由于不知道最后OJ会给多大的测试，因此这里暂且分配256x256的矩阵，到时候详细OJ描述下来之后还要修改
    //------------------DEFINE-----------------------------------
    int M, N, P, K;
    cin >> M >> N >> P >> K;
    int* denseMatrix, *result, *sparseMatrixVal;
    int* sparseMatrixIdx;

    const int TPB = 256;
    const int BPG = (K + TPB - 1) / TPB;

    //------------------INPUT-----------------------------------
    denseMatrix = new int[M * N];
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cin >> denseMatrix[i * N + j];
        }
    }

    sparseMatrixIdx = new int[K * 2];
    sparseMatrixVal = new int[K];
    for (int i = 0; i < K; i++) {
        int row, col;
        int value;
        cin >> row >> col >> value;
        sparseMatrixIdx[i * 2] = row;
        sparseMatrixIdx[i * 2 + 1] = col;
        sparseMatrixVal[i] = value;
    }

    
    result = new int[M * P];
    //------------------CUDA-----------------------------------
    int* d_denseMatrix, * d_sparseMatrixVal, * d_result;
    int* d_sparseMatrixIdx;
    hipMalloc((void**)&d_denseMatrix, sizeof(int) * M * N);
    hipMalloc((void**)&d_sparseMatrixIdx, sizeof(int) * K * 2);
    hipMalloc((void**)&d_sparseMatrixVal, sizeof(int) * K);
    hipMalloc((void**)&d_result, sizeof(int) * M * P);
    hipMemcpy(d_denseMatrix, denseMatrix, sizeof(int) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_sparseMatrixIdx, sparseMatrixIdx, sizeof(int) * K * 2, hipMemcpyHostToDevice);
    hipMemcpy(d_sparseMatrixVal, sparseMatrixVal, sizeof(int) * K, hipMemcpyHostToDevice);

    dim3 threadPerBlock = dim3(TPB);
    dim3 blockPerGrid = dim3(BPG);
    smm <<<BPG, TPB >>> (d_denseMatrix, d_sparseMatrixIdx, d_sparseMatrixVal, d_result, M, N, P, K);
    hipMemcpy(result, d_result, sizeof(int) * M * P, hipMemcpyDeviceToHost);
    //------------------OUTPUT-----------------------------------
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++) {
            cout << result[i * P + j] << " ";
        }
        cout << endl;
    }
    //------------------FREE-----------------------------------
    hipFree(d_denseMatrix);
    hipFree(d_sparseMatrixIdx);
    hipFree(d_sparseMatrixVal);
    hipFree(d_result);
    return 0;
}