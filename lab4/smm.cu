#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <string>
#include <stdlib.h>
#include <time.h>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include ""

using namespace std;

__global__ void smm(float* denseMatrix, int* sparseMatrixIdx, float* sparseMatrixVal, float* result, int M, int N, int P, int K) {
    int idx = blockIdx.x;
    if (idx > K) return;
    else {
        int row = sparseMatrixIdx[idx * 2];
        int col = sparseMatrixIdx[idx * 2 + 1];
        float value = sparseMatrixVal[idx];
        int threadId = threadIdx.x;
        if (threadId < N) {
            atomicAdd(&result[threadId * P + col], denseMatrix[threadId * N + row] * value);
        }
    }
    __syncthreads();
    //---------------低配版加速----------------------------------
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx < K) {
    //     printf("K: %d\n", K);
    //     for (int i = 0; i < K; i++) {
    //         printf("sparseMatrixIdx[%d][0]: %d\n", i, sparseMatrixIdx[i * 2]);
    //         printf("sparseMatrixIdx[%d][1]: %d\n", i, sparseMatrixIdx[i * 2 + 1]);
    //         printf("sparseMatrixVal[%d]: %f\n", i, sparseMatrixVal[i]);
    //     }
    //     printf("idx: %d\n", idx);
    //     int row = sparseMatrixIdx[idx * 2];
    //     int col = sparseMatrixIdx[idx * 2 + 1];
    //     float value = sparseMatrixVal[idx];
    //     printf("row: %d, col: %d, value: %f\n", row, col, value);
    //     for (int i = 0; i < N; i++) {
    //         atomicAdd(&result[i * P + col], denseMatrix[i * N + row] * value);
    //     }
    // }
}

int main() {
    srand(time(NULL));
    // 这里由于不知道最后OJ会给多大的测试，因此这里暂且分配256x256的矩阵，到时候详细OJ描述下来之后还要修改
    //------------------DEFINE-----------------------------------
    const int TPB = 1024;
    const int BPG = 64;
    const int DATA_SIZE = TPB * BPG;

    int M, N, P, K;
    cin >> M >> N >> P >> K;
    float* denseMatrix, *result, *sparseMatrixVal;
    int* sparseMatrixIdx;

    //------------------INPUT-----------------------------------
    denseMatrix = new float[M * N];
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cin >> denseMatrix[i * N + j];
        }
    }

    sparseMatrixIdx = new int[K * 2];
    sparseMatrixVal = new float[K];
    for (int i = 0; i < K; i++) {
        int row, col;
        float value;
        cin >> row >> col >> value;
        sparseMatrixIdx[i * 2] = row;
        sparseMatrixIdx[i * 2 + 1] = col;
        sparseMatrixVal[i] = value;
    }

    
    result = new float[M * P];
    //------------------CUDA-----------------------------------
    float* d_denseMatrix, * d_sparseMatrixVal, * d_result;
    int* d_sparseMatrixIdx;
    hipMalloc((void**)&d_denseMatrix, sizeof(float) * M * N);
    hipMalloc((void**)&d_sparseMatrixIdx, sizeof(int) * K * 2);
    hipMalloc((void**)&d_sparseMatrixVal, sizeof(float) * K);
    hipMalloc((void**)&d_result, sizeof(float) * M * P);
    hipMemcpy(d_denseMatrix, denseMatrix, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_sparseMatrixIdx, sparseMatrixIdx, sizeof(int) * K * 2, hipMemcpyHostToDevice);
    hipMemcpy(d_sparseMatrixVal, sparseMatrixVal, sizeof(float) * K, hipMemcpyHostToDevice);

    dim3 threadPerBlock = dim3(TPB);
    dim3 blockPerGrid = dim3((DATA_SIZE + TPB - 1) / TPB);
    smm << <BPG, TPB >> > (d_denseMatrix, d_sparseMatrixIdx, d_sparseMatrixVal, d_result, M, N, P, K);
    hipMemcpy(result, d_result, sizeof(float) * M * P, hipMemcpyDeviceToHost);
    //------------------OUTPUT-----------------------------------
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++) {
            cout << result[i * P + j] << " ";
        }
        cout << endl;
    }
    //------------------FREE-----------------------------------
    hipFree(d_denseMatrix);
    hipFree(d_sparseMatrixIdx);
    hipFree(d_sparseMatrixVal);
    hipFree(d_result);
    return 0;
}
/*
4 4 4 5
2.32 3.2 6.0 7.2
0.9 3.5 2.0 3.1
2.1 3.2 0.6 9.2
3.2 0.9 3.2 5.3
0 2 0.3
1 0 6.3
2 3 0.6
3 1 0.6
3 2 2.3
*/